
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>

#define T 1024 // numero max de threads por bloco

//GPU
__global__ void bubbleSort_cuda(int *vet, int temp, int n){
    int j =  2*(blockIdx.x * blockDim.x + threadIdx.x);

        if (vet[j] > vet[j+1] && j<n-1){
            //swap(&vet[j],&vet[j+1]);
            temp=vet[j];
            vet[j]=vet[j+1];
            vet[j+1]=temp;
        }

}
//GPU
__global__ void bubbleSort_cuda2(int *vet, int temp, int n){
    int j =  2*(blockIdx.x * blockDim.x + threadIdx.x)+1;

        if (vet[j] > vet[j+1] && j<n-2){
            //swap(&vet[j],&vet[j+1]);
            temp=vet[j];
            vet[j]=vet[j+1];
            vet[j+1]=temp;
        }

}

double wtime() {
  struct timespec t;
  clock_gettime(CLOCK_REALTIME, &t);
  return t.tv_sec + (double) t.tv_nsec / 1000000000;
}

// CPU: função principal
int main(int argc, char const *argv[]) {

    char filename[100];

    int i;
    char aux[100];

    sprintf (filename, "%s", argv[1]); //escreve o nome do arquivo passado

    //pega o numero de elementos do nome do arquivo/////
    for(i=0;i<100;i++){
        if(filename[i]!='_'){
            aux[i]=filename[i];
        }
        else
            break;
    }
    int n_elementos = atoi(aux);


    int *vet_original; //cria vetor original
    int *vet_result; //cria vetor para operacoes
    int size = sizeof(int)*n_elementos;

    hipHostMalloc((void **) &vet_original, size, hipHostMallocDefault);
    hipHostMalloc((void **) &vet_result, size, hipHostMallocDefault);

    FILE* arquivo;
    arquivo = fopen(filename,"r");

    for(i = 0; i < n_elementos; i++){
        fscanf (arquivo, "%d", &vet_original[i]); //pega os valores do arquivo e salva no vetor na cpu
    }
    fclose(arquivo);


    int *vet_cuda;
    hipMalloc((void **) &vet_cuda, size);

    int temp;
    hipMalloc((void **) &temp, sizeof(int));


    hipMemcpy(vet_cuda, vet_original, size, hipMemcpyHostToDevice); //copia o vetor da cpu para o vetor da gpu

    double start_time, end_time;
    start_time=wtime();
    for(i=0;i<n_elementos;i++){
        bubbleSort_cuda<<<(int)ceil(n_elementos/T),T>>>(vet_cuda, temp, n_elementos);
        bubbleSort_cuda2<<<(int)ceil(n_elementos/T),T>>>(vet_cuda, temp, n_elementos);
    }
    end_time=wtime();

    printf("\nTempo de ordenacao CUDA: %f\n", (end_time-start_time));

    hipMemcpy(vet_result, vet_cuda, size, hipMemcpyDeviceToHost); //copia o vetor da gpu para o vetor da cpu


    sprintf (filename, "%d_vet_ordenado_cuda.result", n_elementos);
    arquivo = fopen(filename,"w");
    for(i=0;i<n_elementos;i++){
        fprintf(arquivo,"%d ", vet_result[i]); //grava o resultado no arquivo de saida
    }
    fclose(arquivo);


    // Libera a Memória Global (GPU)
    hipFree(vet_cuda);

    // Libera a Memória Global (CPU)
    hipHostFree(vet_result);
    hipHostFree(vet_original);


  return 0;
}
